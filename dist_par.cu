
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Macro para checagem de erro das chamadas às funções do CUDA
#define checa_cuda(result) \
    if (result != hipSuccess) { \
        printf("%s\n", hipGetErrorString(result)); \
        exit(1); \
    }

char *aloca_sequencia(int n) {
    char *seq;

    seq = (char *) malloc((n + 1) * sizeof(char));
    if (seq == NULL) {
        printf("\nErro na alocação de estruturas\n");
        exit(1);
    }
    return seq;
}

__global__ void inicializa_GPU(int nLinhas, int mColunas, int *a)
{
    int i; // id GLOBAL da thread

    i = blockIdx.x * blockDim.x + threadIdx.x;

    // Inicializa as colunas da 1ª linha
    if (i < mColunas) {
        a[i] = i;
    }

    // Inicializa a 1ª coluna
    if (i < nLinhas) {
       a[i * mColunas] = i;
    }
}

// Kernel executado na GPU por todas as threads de todos os blocos
__global__ void distancia_GPU(int nLinhas, int mColunas, int *a, char *s, char *r, int n, int m, int *d, int deslocamento, int deslocamentoS)
{
    int i; // id GLOBAL da thread

    i = blockIdx.x * blockDim.x + threadIdx.x;
    
    int min, celulaDiagonal;

    int it = 0;  // Usado para andar com o índice de r
    int rodada = 0;

    while(rodada < nLinhas + mColunas - 3 && deslocamentoS + i + 1 < nLinhas) {

        // Se a thread estiver após a 1ª coluna  E  no máximo na última coluna  E  no máximo na última linha
        if (rodada - i >= 0  && rodada - i < mColunas - 1 && i + 1 < nLinhas) {
            
            // Se s[i+1] e r[it+1] forem iguais, copia o valor da diagonal; senão, copia o valor da diagonal acrescido de uma unidade
            celulaDiagonal = s[deslocamentoS + i+1] == r[it + 1] ? a[deslocamento + i*mColunas - i + rodada] : a[deslocamento + i*mColunas - i + rodada] + 1;

            // Mínimo entre a célula diagonal (já calculada) e a célula de cima (acrescida de uma unidade)
            min = celulaDiagonal < a[deslocamento + i*mColunas - i + rodada + 1] + 1 ? celulaDiagonal : a[deslocamento + i*mColunas - i + rodada + 1] + 1;

            // Mínimo entre a célula à esquerda e o mínimo anterior
            if (a[deslocamento + i*mColunas + mColunas + 1 - i + rodada - 1] + 1 < min) {
                a[deslocamento + i*mColunas + mColunas + 1 - i + rodada] = a[deslocamento + i*mColunas + mColunas + 1 - i + rodada - 1] + 1;
            } else {
                a[deslocamento + i*mColunas + mColunas + 1 - i + rodada] = min;
            }

            it++;
        }

        rodada++;

        // Sincronização de barreira entre todas as threads do BLOCO
        __syncthreads();
    }

    if (i == 0) {
        *d = a[nLinhas * mColunas - 1];
    }
}

// Programa principal
int main(int argc, char **argv) {
    int nLinhas,
    mColunas,
    nBytes,
    *d_a,    // Vetor (matriz de distância) da GPU (device)
    
    *d_dist, // Variável da GPU (device) que conterá a última célula da matriz
    h_dist;  // Valor de retorno da última célula da matriz (conterá a distância)
    
    const int N_THREADS_BLOCO = 1024;

    int n,  // Tamanho da sequência s
        m;  // Tamanho da sequência r

    char *h_s,  // Sequência s de entrada (vetor com tamanho n+1)
         *h_r,  // Sequência r de entrada (vetor com tamanho m+1)
         *d_s,
         *d_r;

    FILE *arqEntrada;  // Arquivo texto de entrada

    if(argc != 2) {
        printf("O programa foi executado com argumentos incorretos.\n");
        printf("Uso: ./dist_seq <nome arquivo entrada>\n");
        exit(1);
    }

    // Abre arquivo de entrada
    arqEntrada = fopen(argv[1], "rt");

    if (arqEntrada == NULL) {
        printf("\nArquivo texto de entrada não encontrado\n");
        exit(1);
    }

    // Lê tamanho das sequências s e r
    fscanf(arqEntrada, "%d %d", &n, &m);
    n++;
    m++;

    nLinhas = n;
    mColunas = m;
    nBytes = nLinhas * mColunas * sizeof(int);

    // Aloca vetores s e r
    h_s = aloca_sequencia(n);
    h_r = aloca_sequencia(m);

    // Lê sequências do arquivo de entrada
    h_s[0] = ' ';
    h_r[0] = ' ';
    fscanf(arqEntrada, "%s", &(h_s[1]));
    fscanf(arqEntrada, "%s", &(h_r[1]));

    // Fecha arquivo de entrada
    fclose(arqEntrada);

    
    /* Alocação de memória e checagem de erro */

    // Aloca vetor (matriz de distância) na memória global da GPU
    checa_cuda(hipMalloc((void **)&d_a, nBytes));

    // Aloca variável (distância) na memória global da GPU
    checa_cuda(hipMalloc((void **)&d_dist, sizeof(int)));
     
    // Aloca vetor (sequência r) na memória global da GPU
    checa_cuda(hipMalloc((void **)&d_r, m*sizeof(char)));

    // Aloca vetor (sequência s) na memória global da GPU
    checa_cuda(hipMalloc((void **)&d_s, n*sizeof(char)));
    
    
    hipEvent_t d_ini, d_fim;
    hipEventCreate(&d_ini);
    hipEventCreate(&d_fim);
    hipEventRecord(d_ini, 0);
    
    // Determina nBlocos em função de mColunas e N_THREADS_BLOCO
    // nBlocos = teto(mColunas / N_THREADS_BLOCO)
    int nBlocos = (mColunas + N_THREADS_BLOCO - 1) / N_THREADS_BLOCO;

    inicializa_GPU<<<nBlocos, N_THREADS_BLOCO>>>(nLinhas, mColunas, d_a);
    
    // Copia a sequência s do host para a GPU e checa se houve erro
    checa_cuda(hipMemcpy(d_s, h_s, n*sizeof(char), hipMemcpyHostToDevice));

    // Copia a sequência r do host para a GPU e checa se houve erro
    checa_cuda(hipMemcpy(d_r, h_r, m*sizeof(char), hipMemcpyHostToDevice));
    
    // Host espera GPU terminar de executar
    hipDeviceSynchronize();

    int deslocamento = 0;   // Deslocamento de x posições/células na matriz. Ou seja, a cada iteração do while abaixo é deslocado 1 bloco de linhas na matriz
    int deslocamentoS = 0;  // Deslocamento do índice de s

    int repete = nLinhas;
    
    while (repete > 0) {
    
        // Calcula a distância de edição na GPU
        distancia_GPU<<<nBlocos, N_THREADS_BLOCO>>>(nLinhas, mColunas, d_a, d_s, d_r, n, m, d_dist, deslocamento, deslocamentoS);
        
        deslocamento += N_THREADS_BLOCO * mColunas;
        deslocamentoS += N_THREADS_BLOCO; // guarda o novo início de s[] (com o deslocamento p/ a próxima chamada do bloco)
        repete = repete - N_THREADS_BLOCO;
    }
    
    checa_cuda(hipMemcpy(&h_dist, d_dist, sizeof(int), hipMemcpyDeviceToHost));
    
    hipEventRecord(d_fim, 0);
    hipEventSynchronize(d_fim);
    float d_tempo;      // Tempo de execução na GPU em milissegundos
    hipEventElapsedTime(&d_tempo, d_ini, d_fim);
    hipEventDestroy(d_ini);
    hipEventDestroy(d_fim);

    printf("%d\n", h_dist);
    printf("%.2f\n", d_tempo);
    
    // Libera vetor (matriz de distância) da memória global da GPU
    hipFree(d_a);

    // Libera vetores da memória global da GPU
    hipFree(d_s);
    hipFree(d_r);

    // Libera variável da memória global da GPU
    hipFree(d_dist);

    // Libera vetores da memória do host
    free(h_s);
    free(h_r);

    return 0;
}